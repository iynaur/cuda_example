#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

// Compute vector sum C = A+B
//CUDA kernel. Each thread performes one pair-wise addition

__global__ void vector_add(float *a, float *b, float *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (100000)
#define THREADS_PER_BLOCK 1000

int main()
{
    float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int size = N * sizeof( float );
	/* allocate space for device copies of a, b, c */
	/* allocate space for host copies of a, b, c and setup input values */

//Allocate memory for each vector on host
	a = (float *)malloc( size );
	b = (float *)malloc( size );
	c = (float *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	vector_add<<< 100, 1000 >>>( d_a, d_b, d_c );

//Synchronize threads
hipDeviceSynchronize();

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %f\n",c[0] );
	printf( "c[%d] = %f\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
